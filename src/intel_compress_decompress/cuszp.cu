#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <papi.h>
#include <cuSZp_utility.h>
#include <cuSZp_entry_f32.h>
#include <cuSZp_entry_f64.h>
#include <cuSZp_timer.h>

#define MAX_ITERATIONS 25
#define CONFIDENCE_LEVEL 1.96
#define MAX_POWERCAP_EVENTS 64
#define CHECK_NVML(call) { nvmlReturn_t result = call; if (result != NVML_SUCCESS) { fprintf(stderr, "NVML Error: %s\n", nvmlErrorString(result)); exit(1); } }
#define CHECK_PAPI(call) { int retval = call; if (retval != PAPI_OK) { fprintf(stderr, "PAPI Error: %s\n", PAPI_strerror(retval)); exit(1); } }

typedef struct {
    double compression_time;
    double decompression_time;
    double compression_throughput;
    double decompression_throughput;
    double compression_ratio;
    double max_error;
    double avg_error;
    double mse;
    double psnr;
    double nrmse;
    unsigned long compressed_size;
    double cpu_comp_energy;
    double cpu_decomp_energy;
    unsigned long gpu_comp_energy;
    unsigned long gpu_decomp_energy;
} CompressionMetrics;

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec / 1e9;
}

double calculate_mean(double *data, int n) {
    double sum = 0.0;
    for (int i = 0; i < n; i++) {
        sum += data[i];
    }
    return sum / n;
}

double calculate_std_dev(double *data, int n, double mean) {
    double sum_squared_diff = 0.0;
    for (int i = 0; i < n; i++) {
        double diff = data[i] - mean;
        sum_squared_diff += diff * diff;
    }
    return sqrt(sum_squared_diff / (n - 1));
}

bool within_confidence_interval(double *data, int n) {
    if (n < 2) return false;
    double mean = calculate_mean(data, n);
    double std_dev = calculate_std_dev(data, n, mean);
    double margin_of_error = CONFIDENCE_LEVEL * (std_dev / sqrt(n));
    double lower_bound = mean - margin_of_error;
    double upper_bound = mean + margin_of_error;

    for (int i = 0; i < n; i++) {
        if (data[i] < lower_bound || data[i] > upper_bound) {
            return false;
        }
    }
    return true;
}

void calculate_error_metrics(void *original, void *decompressed, size_t num_elements, int data_type, CompressionMetrics *metrics) {
    double max_error = 0.0, sum_squared_error = 0.0, sum_error = 0.0;
    double min_val = INFINITY, max_val = -INFINITY;

    if (data_type == 0) { // float
        float *orig = (float *)original;
        float *decomp = (float *)decompressed;
        for (size_t i = 0; i < num_elements; i++) {
            double error = fabs(orig[i] - decomp[i]);
            max_error = fmax(max_error, error);
            sum_squared_error += error * error;
            sum_error += error;
            min_val = fmin(min_val, orig[i]);
            max_val = fmax(max_val, orig[i]);
        }
    } else { // double
        double *orig = (double *)original;
        double *decomp = (double *)decompressed;
        for (size_t i = 0; i < num_elements; i++) {
            double error = fabs(orig[i] - decomp[i]);
            max_error = fmax(max_error, error);
            sum_squared_error += error * error;
            sum_error += error;
            min_val = fmin(min_val, orig[i]);
            max_val = fmax(max_val, orig[i]);
        }
    }

    metrics->max_error = max_error;
    metrics->avg_error = sum_error / num_elements;
    metrics->mse = sum_squared_error / num_elements;
    double value_range = max_val - min_val;
    metrics->psnr = 20 * log10(value_range) - 10 * log10(metrics->mse);
    metrics->nrmse = sqrt(metrics->mse) / value_range;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <dataset_file> <error_mode> <error_bound>\n", argv[0]);
        return 1;
    }

    const char *dataset_file = argv[1];
    const char *error_mode = argv[2];
    double error_bound = atof(argv[3]);

    // Initialize CUDA
    hipSetDevice(0);

    // Initialize NVML
    nvmlInit();
    nvmlDevice_t device;
    CHECK_NVML(nvmlDeviceGetHandleByIndex(0, &device));

    // Initialize PAPI
    int EventSet = PAPI_NULL;
    long long values[MAX_POWERCAP_EVENTS];
    int num_events = 0;
    char event_names[MAX_POWERCAP_EVENTS][PAPI_MAX_STR_LEN];
    int data_type[MAX_POWERCAP_EVENTS];

    CHECK_PAPI(PAPI_library_init(PAPI_VER_CURRENT));
    CHECK_PAPI(PAPI_create_eventset(&EventSet));

    // Find and add powercap events
    int code = PAPI_NATIVE_MASK;
    PAPI_event_info_t info;
    while (PAPI_enum_cmp_event(&code, PAPI_ENUM_FIRST, 0) == PAPI_OK) {
        if (PAPI_get_event_info(code, &info) == PAPI_OK) {
            if (strstr(info.symbol, "powercap")) {
                if (PAPI_add_event(EventSet, code) == PAPI_OK) {
                    strncpy(event_names[num_events], info.symbol, PAPI_MAX_STR_LEN);
                    data_type[num_events] = info.data_type;
                    num_events++;
                }
            }
        }
    }

    // Read dataset
    size_t num_elements;
    int status = 0;
    void *data = NULL;
    int data_type;

    if (strstr(dataset_file, "nyx") != NULL || strstr(dataset_file, "hacc") != NULL || strstr(dataset_file, "miranda") != NULL) {
        data = (void *)readFloatData_Yafan(dataset_file, &num_elements, &status);
        data_type = 0; // float
    } else if (strstr(dataset_file, "s3d") != NULL) {
        data = (void *)readDoubleData_Yafan(dataset_file, &num_elements, &status);
        data_type = 1; // double
    } else {
        fprintf(stderr, "Unknown dataset %s\n", dataset_file);
        return 1;
    }

    if (status != 0) {
        fprintf(stderr, "Failed to read dataset %s\n", dataset_file);
        return 1;
    }

    // Allocate memory on GPU
    void *d_data, *d_compressed, *d_decompressed;
    size_t data_size = num_elements * (data_type == 0 ? sizeof(float) : sizeof(double));
    hipMalloc(&d_data, data_size);
    hipMalloc(&d_compressed, data_size);
    hipMalloc(&d_decompressed, data_size);
    hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice);

    // Prepare for compression
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Metrics arrays
    double compression_times[MAX_ITERATIONS];
    double decompression_times[MAX_ITERATIONS];
    CompressionMetrics metrics[MAX_ITERATIONS];

    int iteration = 0;
    bool confidence_interval_reached = false;

    while (iteration < MAX_ITERATIONS && !confidence_interval_reached) {
        // Compression
        size_t compressed_size;
        
        // Start CPU and GPU energy measurement for compression
        unsigned long long gpu_energy_start, gpu_energy_end;
        CHECK_PAPI(PAPI_start(EventSet));
        CHECK_NVML(nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_start));

        double start_time = get_time();
        if (data_type == 0) {
            SZp_compress_deviceptr_f32((float *)d_data, (unsigned char *)d_compressed, num_elements, &compressed_size, error_bound, stream);
        } else {
            SZp_compress_deviceptr_f64((double *)d_data, (unsigned char *)d_compressed, num_elements, &compressed_size, error_bound, stream);
        }
        hipStreamSynchronize(stream);
        double end_time = get_time();
        
        CHECK_NVML(nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_end));
        CHECK_PAPI(PAPI_stop(EventSet, values));

        compression_times[iteration] = end_time - start_time;
        metrics[iteration].gpu_comp_energy = gpu_energy_end - gpu_energy_start;

        // Calculate CPU energy for compression
        metrics[iteration].cpu_comp_energy = 0.0;
        for (int i = 0; i < num_events; i++) {
            if (strstr(event_names[i], "ENERGY_UJ") && data_type[i] == PAPI_DATATYPE_UINT64) {
                metrics[iteration].cpu_comp_energy += values[i] / 1.0e6;
            }
        }

        // Decompression
        // Start CPU and GPU energy measurement for decompression
        CHECK_PAPI(PAPI_start(EventSet));
        CHECK_NVML(nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_start));

        start_time = get_time();
        if (data_type == 0) {
            SZp_decompress_deviceptr_f32((float *)d_decompressed, (unsigned char *)d_compressed, num_elements, compressed_size, error_bound, stream);
        } else {
            SZp_decompress_deviceptr_f64((double *)d_decompressed, (unsigned char *)d_compressed, num_elements, compressed_size, error_bound, stream);
        }
        hipStreamSynchronize(stream);
        end_time = get_time();

        CHECK_NVML(nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_end));
        CHECK_PAPI(PAPI_stop(EventSet, values));

        decompression_times[iteration] = end_time - start_time;
        metrics[iteration].gpu_decomp_energy = gpu_energy_end - gpu_energy_start;

        // Calculate CPU energy for decompression
        metrics[iteration].cpu_decomp_energy = 0.0;
        for (int i = 0; i < num_events; i++) {
            if (strstr(event_names[i], "ENERGY_UJ") && data_type[i] == PAPI_DATATYPE_UINT64) {
                metrics[iteration].cpu_decomp_energy += values[i] / 1.0e6;
            }
        }

        // Calculate metrics
        void *h_decompressed = malloc(data_size);
        hipMemcpy(h_decompressed, d_decompressed, data_size, hipMemcpyDeviceToHost);

        calculate_error_metrics(data, h_decompressed, num_elements, data_type, &metrics[iteration]);

        metrics[iteration].compression_time = compression_times[iteration];
        metrics[iteration].decompression_time = decompression_times[iteration];
        metrics[iteration].compression_throughput = (data_size / 1e9) / compression_times[iteration];
        metrics[iteration].decompression_throughput = (data_size / 1e9) / decompression_times[iteration];
        metrics[iteration].compression_ratio = (double)data_size / compressed_size;
        metrics[iteration].compressed_size = compressed_size;

        free(h_decompressed);

        // Write metrics to CSV file
        FILE *csv_file = fopen("cuszip_compression_metrics.csv", "a");
        if (csv_file == NULL) {
            fprintf(stderr, "Error opening CSV file\n");
        } else {
            fprintf(csv_file, "cuSZp,%s,%s,%e,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%lu,%f,%f,%lu,%lu\n",
                    dataset_file, error_mode, error_bound, iteration,
                    metrics[iteration].compression_time,
                    metrics[iteration].decompression_time,
                    metrics[iteration].compression_throughput,
                    metrics[iteration].decompression_throughput,
                    metrics[iteration].compression_ratio,
                    metrics[iteration].max_error,
                    metrics[iteration].avg_error,
                    metrics[iteration].mse,
                    metrics[iteration].psnr,
                    metrics[iteration].nrmse,
                    metrics[iteration].compressed_size,
                    metrics[iteration].cpu_comp_energy,
                    metrics[iteration].cpu_decomp_energy,
                    metrics[iteration].gpu_comp_energy,
                    metrics[iteration].gpu_decomp_energy);
            fclose(csv_file);
        }

        iteration++;

        // Check if we've reached the confidence interval
        if (iteration >= 5) {
            confidence_interval_reached =
                within_confidence_interval(compression_times, iteration) &&
                within_confidence_interval(decompression_times, iteration);
        }
    }
        
    free(data);
    hipFree(d_data);
    hipFree(d_compressed);
    hipFree(d_decompressed);
    hipStreamDestroy(stream);
    nvmlShutdown();
    PAPI_cleanup_eventset(EventSet);
    PAPI_destroy_eventset(&EventSet);
    PAPI_shutdown();

}

