#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <papi.h>
#include <cuSZp_utility.h>
#include <cuSZp_entry_f32.h>
#include <cuSZp_entry_f64.h>
#include <cuSZp_timer.h>

#define MAX_ITERATIONS 25
#define CONFIDENCE_LEVEL 1.96
#define MAX_POWERCAP_EVENTS 64
#define CHECK_NVML(call) { nvmlReturn_t result = call; if (result != NVML_SUCCESS) { fprintf(stderr, "NVML Error: %s\n", nvmlErrorString(result)); exit(1); } }
#define CHECK_CUDA(call) { hipError_t error = call; if (error != hipSuccess) { fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error)); exit(1); } }

typedef struct {
    double compression_time;
    double decompression_time;
    double compression_throughput;
    double decompression_throughput;
    double compression_ratio;
    double max_error;
    double avg_error;
    double mse;
    double psnr;
    double nrmse;
    unsigned long compressed_size;
    double cpu_comp_energy;
    double cpu_decomp_energy;
    unsigned long gpu_comp_energy;
    unsigned long gpu_decomp_energy;
} CompressionMetrics;

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec / 1e9;
}

double calculate_mean(double *data, int n) {
    double sum = 0.0;
    for (int i = 0; i < n; i++) {
        sum += data[i];
    }
    return sum / n;
}

double calculate_std_dev(double *data, int n, double mean) {
    double sum_squared_diff = 0.0;
    for (int i = 0; i < n; i++) {
        double diff = data[i] - mean;
        sum_squared_diff += diff * diff;
    }
    return sqrt(sum_squared_diff / (n - 1));
}

bool within_confidence_interval(double *data, int n) {
    if (n < 2) return false;
    double mean = calculate_mean(data, n);
    double std_dev = calculate_std_dev(data, n, mean);
    double margin_of_error = CONFIDENCE_LEVEL * (std_dev / sqrt(n));
    double lower_bound = mean - margin_of_error;
    double upper_bound = mean + margin_of_error;

    for (int i = 0; i < n; i++) {
        if (data[i] < lower_bound || data[i] > upper_bound) {
            return false;
        }
    }
    return true;
}

void calculate_error_metrics(void *original, void *decompressed, size_t num_elements, int data_type, CompressionMetrics *metrics) {
    double max_error = 0.0, sum_squared_error = 0.0, sum_error = 0.0;
    double min_val = INFINITY, max_val = -INFINITY;

    if (data_type == 0) { // float
        float *orig = (float *)original;
        float *decomp = (float *)decompressed;
        for (size_t i = 0; i < num_elements; i++) {
            double error = fabs(orig[i] - decomp[i]);
            max_error = fmax(max_error, error);
            sum_squared_error += error * error;
            sum_error += error;
            min_val = fmin(min_val, orig[i]);
            max_val = fmax(max_val, orig[i]);
        }
    } else { // double
        double *orig = (double *)original;
        double *decomp = (double *)decompressed;
        for (size_t i = 0; i < num_elements; i++) {
            double error = fabs(orig[i] - decomp[i]);
            max_error = fmax(max_error, error);
            sum_squared_error += error * error;
            sum_error += error;
            min_val = fmin(min_val, orig[i]);
            max_val = fmax(max_val, orig[i]);
        }
    }

    metrics->max_error = max_error;
    metrics->avg_error = sum_error / num_elements;
    metrics->mse = sum_squared_error / num_elements;
    double value_range = max_val - min_val;
    metrics->psnr = 20 * log10(value_range) - 10 * log10(metrics->mse);
    metrics->nrmse = sqrt(metrics->mse) / value_range;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <dataset_file> <error_mode> <error_bound>\n", argv[0]);
        return 1;
    }

    char *dataset_file = argv[1];
    char *error_mode = argv[2];
    double error_bound = atof(argv[3]);

    // Initialize CUDA
    CHECK_CUDA(hipSetDevice(0));

    nvmlReturn_t nvml_result = nvmlInit();
    if (nvml_result != NVML_SUCCESS) {
        fprintf(stderr, "Failed to initialize NVML: %s\n", nvmlErrorString(nvml_result));
        // Continue execution, but GPU energy measurements will be disabled
    }

    nvmlDevice_t device;
    unsigned int device_count;
    if (nvmlDeviceGetCount(&device_count) == NVML_SUCCESS && device_count > 0) {
        nvml_result = nvmlDeviceGetHandleByIndex(0, &device);
        if (nvml_result != NVML_SUCCESS) {
            fprintf(stderr, "Failed to get device handle: %s\n", nvmlErrorString(nvml_result));
            // Continue execution, but GPU energy measurements will be disabled
        }
    } else {
        fprintf(stderr, "No NVIDIA devices found or failed to get device count\n");
        // Continue execution, but GPU energy measurements will be disabled
    }

    // Initialize PAPI
    int EventSet = PAPI_NULL;
    long long values[MAX_POWERCAP_EVENTS];
    int num_events = 0;
    char event_names[MAX_POWERCAP_EVENTS][PAPI_MAX_STR_LEN];
    int data_type[MAX_POWERCAP_EVENTS];

    if (PAPI_library_init(PAPI_VER_CURRENT) != PAPI_VER_CURRENT) {
        fprintf(stderr, "PAPI library init error!\n");
        exit(1);
    }

    // Find powercap component
    int numcmp = PAPI_num_components();
    int cid, powercap_cid = -1;
    const PAPI_component_info_t *cmpinfo = NULL;
    for (cid = 0; cid < numcmp; cid++) {
        cmpinfo = PAPI_get_component_info(cid);
        if (cmpinfo == NULL) {
            fprintf(stderr, "PAPI: Failed to get component info\n");
            exit(1);
        }
        if (strstr(cmpinfo->name, "powercap")) {
            powercap_cid = cid;
            break;
        }
    }
    if (cid == numcmp) {
        fprintf(stderr, "PAPI: Failed to find powercap component\n");
        exit(1);
    }

    // Create EventSet
    if (PAPI_create_eventset(&EventSet) != PAPI_OK) {
        fprintf(stderr, "PAPI: Failed to create event set\n");
        exit(1);
    }

    // Find and add powercap events
    int code = PAPI_NATIVE_MASK;
    PAPI_event_info_t info;
    int r = PAPI_enum_cmp_event(&code, PAPI_ENUM_FIRST, powercap_cid);
    while (r == PAPI_OK && num_events < MAX_POWERCAP_EVENTS) {
        if (PAPI_get_event_info(code, &info) == PAPI_OK) {
            if (PAPI_add_event(EventSet, code) == PAPI_OK) {
                strncpy(event_names[num_events], info.symbol, PAPI_MAX_STR_LEN);
                data_type[num_events] = info.data_type;
                num_events++;
            }
        }
        r = PAPI_enum_cmp_event(&code, PAPI_ENUM_EVENTS, powercap_cid);
    }

    // Read dataset
    size_t num_elements;
    int status = 0;
    void *data = NULL;
    int comp_data_type;

    if (strstr(dataset_file, "nyx") != NULL || strstr(dataset_file, "hacc") != NULL || strstr(dataset_file, "miranda") != NULL) {
        data = (void *)readFloatData_Yafan(dataset_file, &num_elements, &status);
        comp_data_type = 0; // float
    } else if (strstr(dataset_file, "s3d") != NULL) {
        data = (void *)readDoubleData_Yafan(dataset_file, &num_elements, &status);
        comp_data_type = 1; // double
    } else {
        fprintf(stderr, "Unknown dataset %s\n", dataset_file);
        return 1;
    }

    if (status != 0) {
        fprintf(stderr, "Failed to read dataset %s\n", dataset_file);
        return 1;
    }

    // Allocate memory on GPU
    void *d_data, *d_compressed, *d_decompressed;
    size_t data_size = num_elements * (comp_data_type == 0 ? sizeof(float) : sizeof(double));
    CHECK_CUDA(hipMalloc(&d_data, data_size));
    CHECK_CUDA(hipMalloc(&d_compressed, data_size));
    CHECK_CUDA(hipMalloc(&d_decompressed, data_size));
    CHECK_CUDA(hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice));

    // Prepare for compression
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    // Metrics arrays
    double compression_times[MAX_ITERATIONS];
    double decompression_times[MAX_ITERATIONS];
    CompressionMetrics metrics[MAX_ITERATIONS];

    int iteration = 0;
    bool confidence_interval_reached = false;

    while (iteration < MAX_ITERATIONS && !confidence_interval_reached) {
        // Compression
        size_t compressed_size;
        
        // Start CPU and GPU energy measurement for compression
        unsigned long long gpu_energy_start = 0, gpu_energy_end = 0;
        if (PAPI_start(EventSet) != PAPI_OK) {
            fprintf(stderr, "PAPI: Failed to start EventSet\n");
            exit(1);
        }
        if (nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_start) != NVML_SUCCESS) {
            fprintf(stderr, "Failed to get initial GPU energy consumption\n");
        }

        double start_time = get_time();
        if (comp_data_type == 0) {
            SZp_compress_deviceptr_f32((float *)d_data, (unsigned char *)d_compressed, num_elements, &compressed_size, error_bound, stream);
        } else {
            SZp_compress_deviceptr_f64((double *)d_data, (unsigned char *)d_compressed, num_elements, &compressed_size, error_bound, stream);
        }
        CHECK_CUDA(hipStreamSynchronize(stream));
        double end_time = get_time();
        
        if (nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_end) != NVML_SUCCESS) {
            fprintf(stderr, "Failed to get initial GPU energy consumption\n");
        }
        if (PAPI_stop(EventSet, values) != PAPI_OK) {
            fprintf(stderr, "PAPI: Failed to stop EventSet\n");
            exit(1);
        }

        compression_times[iteration] = end_time - start_time;
        metrics[iteration].gpu_comp_energy =(gpu_energy_end > gpu_energy_start) ? (gpu_energy_end - gpu_energy_start) : 0;

        // Calculate CPU energy for compression
        metrics[iteration].cpu_comp_energy = 0.0;
        for (int i = 0; i < num_events; i++) {
            if (strstr(event_names[i], "ENERGY_UJ") && data_type[i] == PAPI_DATATYPE_UINT64) {
                metrics[iteration].cpu_comp_energy += values[i] / 1.0e6;
            }
        }

        // Decompression
        // Start CPU and GPU energy measurement for decompression
        if (PAPI_start(EventSet) != PAPI_OK) {
            fprintf(stderr, "PAPI: Failed to start EventSet\n");
            exit(1);
        }
         if (nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_start) != NVML_SUCCESS) {
            fprintf(stderr, "Failed to get initial GPU energy consumption\n");
        }

        start_time = get_time();
        if (comp_data_type == 0) {
            SZp_decompress_deviceptr_f32((float *)d_decompressed, (unsigned char *)d_compressed, num_elements, compressed_size, error_bound, stream);
        } else {
            SZp_decompress_deviceptr_f64((double *)d_decompressed, (unsigned char *)d_compressed, num_elements, compressed_size, error_bound, stream);
        }
        CHECK_CUDA(hipStreamSynchronize(stream));
        end_time = get_time();

        if (nvmlDeviceGetTotalEnergyConsumption(device, &gpu_energy_end) != NVML_SUCCESS) {
            fprintf(stderr, "Failed to get final GPU energy consumption\n");
        }
        if (PAPI_stop(EventSet, values) != PAPI_OK) {
            fprintf(stderr, "PAPI: Failed to stop EventSet\n");
            exit(1);
        }

        decompression_times[iteration] = end_time - start_time;
        metrics[iteration].gpu_decomp_energy = (gpu_energy_end > gpu_energy_start) ? (gpu_energy_end - gpu_energy_start) : 0;

        // Calculate CPU energy for decompression
        metrics[iteration].cpu_decomp_energy = 0.0;
        for (int i = 0; i < num_events; i++) {
            if (strstr(event_names[i], "ENERGY_UJ") && data_type[i] == PAPI_DATATYPE_UINT64) {
                metrics[iteration].cpu_decomp_energy += values[i] / 1.0e6;
            }
        }

        // Calculate metrics
        void *h_decompressed = malloc(data_size);
        CHECK_CUDA(hipMemcpy(h_decompressed, d_decompressed, data_size, hipMemcpyDeviceToHost));

        calculate_error_metrics(data, h_decompressed, num_elements, comp_data_type, &metrics[iteration]);

        metrics[iteration].compression_time = compression_times[iteration];
        metrics[iteration].decompression_time = decompression_times[iteration];
        metrics[iteration].compression_throughput = (data_size / 1e9) / compression_times[iteration];
        metrics[iteration].decompression_throughput = (data_size / 1e9) / decompression_times[iteration];
        metrics[iteration].compression_ratio = (double)data_size / compressed_size;
        metrics[iteration].compressed_size = compressed_size;

        free(h_decompressed);
        // Write metrics to CSV file
        FILE *csv_file = fopen("cuszp_compression_metrics.csv", "a");
        if (csv_file == NULL) {
            fprintf(stderr, "Error opening CSV file\n");
        } else {
            fprintf(csv_file, "cuSZp,%s,%s,%e,%d,%e,%e,%e,%e,%e,%e,%e,%e,%e,%e,%lu,%e,%e,%lu,%lu\n",
                    dataset_file,
                    error_mode,
                    error_bound,
                    iteration,
                    metrics[iteration].compression_time,
                    metrics[iteration].decompression_time,
                    metrics[iteration].compression_throughput,
                    metrics[iteration].decompression_throughput,
                    metrics[iteration].compression_ratio,
                    metrics[iteration].max_error,
                    metrics[iteration].avg_error,
                    metrics[iteration].mse,
                    metrics[iteration].psnr,
                    metrics[iteration].nrmse,
                    metrics[iteration].compressed_size,
                    metrics[iteration].cpu_comp_energy,
                    metrics[iteration].cpu_decomp_energy,
                    metrics[iteration].gpu_comp_energy,
                    metrics[iteration].gpu_decomp_energy);
            fclose(csv_file);
        }

        iteration++;

        // Check if we've reached the confidence interval
        if (iteration >= 5) {
            confidence_interval_reached =
                within_confidence_interval(compression_times, iteration) &&
                within_confidence_interval(decompression_times, iteration);
        }
    }
        
    // Clean up
    free(data);
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipFree(d_compressed));
    CHECK_CUDA(hipFree(d_decompressed));
    CHECK_CUDA(hipStreamDestroy(stream));
    CHECK_NVML(nvmlShutdown());

    // PAPI cleanup
    if (PAPI_cleanup_eventset(EventSet) != PAPI_OK) {
        fprintf(stderr, "PAPI: Failed to cleanup EventSet\n");
    }
    if (PAPI_destroy_eventset(&EventSet) != PAPI_OK) {
        fprintf(stderr, "PAPI: Failed to destroy EventSet\n");
    }
    PAPI_shutdown();

    return 0;
}
